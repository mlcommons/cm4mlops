#include "hip/hip_runtime.h"
#ifndef WINDOWS
 #include <unistd.h>
#endif

#include <stdio.h>
#include <hip/hip_runtime.h>

int main(int argc, char *argv[])
{
  int ndev=0;
  int id=0;
  hipError_t error;
  hipDeviceProp_t features;

  int rtver=0;
  int dver=0;

  /* Get number of devices */
  error = hipGetDeviceCount(&ndev);
  if (error != hipSuccess) {
    printf("Error: problem obtaining number of CUDA devices: %d\n", error);
    return 1;
  }

  /* Iterating over devices */
  for (id=0; id<ndev; id++)
  {
     hipSetDevice(id);

     printf("GPU Device ID: %d\n", id);

     hipGetDeviceProperties(&features, id);
     if (error != hipSuccess) {
       printf("Error: problem obtaining features of CUDA devices: %d\n", error);
       return 1;
     }

     printf("GPU Name: %s\n", features.name);
     printf("GPU compute capability: %d.%d\n", features.major, features.minor);

     error=hipDriverGetVersion(&dver);
     if (error != hipSuccess) {
       printf("Error: problem obtaining CUDA driver version: %d\n", error);
       return 1;
     }

     error=hipRuntimeGetVersion(&rtver);
     if (error != hipSuccess) {
       printf("Error: problem obtaining CUDA run-time version: %d\n", error);
       return 1;
     }

     printf("CUDA driver version: %d.%d\n", dver/1000, (dver%100)/10);
     printf("CUDA runtime version: %d.%d\n", rtver/1000, (rtver%100)/10);

     printf("Global memory: %llu\n", (unsigned long long) features.totalGlobalMem);
     printf("Max clock rate: %f MHz\n", features.clockRate * 0.001);

     printf("Total amount of shared memory per block: %lu\n", features.sharedMemPerBlock);
     printf("Total number of registers available per block: %d\n", features.regsPerBlock);
     printf("Warp size: %d\n", features.warpSize);
     printf("Maximum number of threads per multiprocessor:  %d\n", features.maxThreadsPerMultiProcessor);
     printf("Maximum number of threads per block: %d\n", features.maxThreadsPerBlock);
     printf("Max dimension size of a thread block X: %d\n", features.maxThreadsDim[0]);
     printf("Max dimension size of a thread block Y: %d\n", features.maxThreadsDim[1]);
     printf("Max dimension size of a thread block Z: %d\n", features.maxThreadsDim[2]);
     printf("Max dimension size of a grid size X: %d\n", features.maxGridSize[0]);
     printf("Max dimension size of a grid size Y: %d\n", features.maxGridSize[1]);
     printf("Max dimension size of a grid size Z: %d\n", features.maxGridSize[2]);
     printf("\n");
  }

  return error;
}
